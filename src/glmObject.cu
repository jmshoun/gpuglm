#include "hip/hip_runtime.h"
#include "glmObject.h"

// Constructors / Destructors /////////////////////////////////////////////////

glmObject::glmObject(glmData *_data, glmFamily *_family,
		glmControl *_control, glmVector<num_t> *_startingBeta) {
	// Construct member objects
	data = _data;
	family = _family;
	control = _control;
	results = new glmResults(_startingBeta);

	// Set common dimensional parameters
	nBeta = results->getNBeta();
	nObs = data->getNObs();

	// Create common vectors need in the workspace
	gradient = new glmVector<num_t>(nBeta, true, true);
	betaDelta = new glmVector<num_t>(nBeta, true, true);
	yDelta = new glmVector<num_t>(nObs, true, true);
	predictions = new glmVector<num_t>(nObs, true, true);

	// Create CUBLAS handle
	CUBLAS_WRAP(hipblasCreate(&handle));

	return;
}

glmObject::~glmObject() {
	delete data;
	delete family;
	delete control;
	delete results;

	delete predictions;
	delete yDelta;
	delete gradient;

	CUBLAS_WRAP(hipblasDestroy(handle));
}

// CUDA Kernels Used by Updating Functions ////////////////////////////////////

__global__ void factorPredictKernel(int n, factor_t *factor, num_t *betas,
		num_t *result) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	factor_t factorValue;

	if (i < n) {
		factorValue = factor[i];
		if (factorValue > 1) {
			result[i] += betas[factorValue];
		}
	}

	return;
}

// Gradient Functions /////////////////////////////////////////////////////////

void glmObject::updateGradient(void) {
	glmVector<num_t> *y = data->getY();
	glmVector<num_t> *weights = data->getWeights();

	this->updatePredictions();

	// Calculate yDelta as y - yHat
	vectorDifference(y, predictions, yDelta);

	// Add weights to yDelta
	if (weights != NULL) {
		vectorMultiply(yDelta, weights, yDelta);
	}

	// Calculate gradient from yDelta * weights...
	this->updateGradientXNumeric();
	this->updateGradientXFactor();
	this->updateGradientIntercept();

	return;
}

void glmObject::updateGradientIntercept(void) {
	num_t *interceptGradientElement = gradient->getDeviceElement(nBeta - 1);
	vectorSum(yDelta, interceptGradientElement);
	return;
}

void glmObject::updateGradientXNumeric(void) {
	glmMatrix<num_t> *xNumeric = data->getXNumeric();

	if (xNumeric != NULL) {
		xNumeric->columnProduct(handle, yDelta, gradient);
	}

	return;
}

void glmObject::updateGradientXFactor(void) {
	if (data->getXFactor() != NULL) {
		for (int i = 0; i < data->getNFactors(); i++) {
			this->updateGradientSingleFactor(i);
		}
	}

	return;
}

void glmObject::updateGradientSingleFactor(int index) {
	int gradientOffset = data->getFactorOffset(index) + 2;
	int factorLength = data->getFactorLength(index);
	glmVector<factor_t> *factorColumn = data->getFactorColumn(index);

	factorProduct(factorColumn, factorLength, yDelta,
			gradient->getDeviceElement(gradientOffset));

	return;
}

// Prediction Functions ///////////////////////////////////////////////////////

void glmObject::updatePredictions(void) {
	linkFunction invLink = family->getInvLink();

	this->updatePredictionXNumeric();
	this->updatePredictionXFactor();
	this->updatePredictionIntercept();

	(*invLink)(predictions, predictions, 0.0);

	return;
}

void glmObject::updatePredictionIntercept(void) {
	glmVector<num_t> *beta = results->getBeta();

	beta->copyDeviceToHost();
	num_t intercept = beta->getHostData()[nBeta - 1];
	vectorAddScalar(predictions, intercept, predictions);
	return;
}

void glmObject::updatePredictionXNumeric(void) {
	glmVector<num_t> *beta = results->getBeta();
	glmMatrix<num_t> *xNumeric = data->getXNumeric();

	if (xNumeric != NULL) {
		xNumeric->rowProduct(handle, beta, predictions);
	}

	return;
}

void glmObject::updatePredictionXFactor(void) {
	if (data->getXFactor() != NULL) {
		for (int i = 0; i < data->getNFactors(); i++) {
			this->updatePredictionSingleFactor(i);
		}
	}

	return;
}

void glmObject::updatePredictionSingleFactor(int index) {
	glmVector<num_t> *beta = results->getBeta();
	linkFunction invLink = family->getInvLink();

	int betaOffset = data->getFactorOffset(index);
	factor_t *factorColumn = data->getRawFactorColumn(index);
	int numBlocks = nObs / THREADS_PER_BLOCK +
			(nObs % THREADS_PER_BLOCK ? 1 : 0);

	factorPredictKernel<<<numBlocks, THREADS_PER_BLOCK>>>(nObs,
			factorColumn, beta->getDeviceData() + betaOffset,
			predictions->getDeviceData());
	CUDA_WRAP(hipPeekAtLastError());

	return;
}
