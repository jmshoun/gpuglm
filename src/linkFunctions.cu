#include "hip/hip_runtime.h"
#include "linkFunctions.h"

#include <iostream>
#include <cstdlib>
#include <cstring>

///////////////////////////////////////////////////////////////////////////////
// Device-side Kernels For Link Functions                                    //
///////////////////////////////////////////////////////////////////////////////

// Link Functions /////////////////////////////////////////////////////////////

__global__ void cudaLogit(int n, num_t *input, num_t *output) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
#ifdef GPUGLM_FASTMATH
		output[i] = __logf(__fdividef(input[i], __fsub_rn(1.0, input[i])));
#else
		output[i] = log(input[i] / (1.0 - input[i]));
#endif
	}
	return;
}
__global__ void cudaLog(int n, num_t *input, num_t *output) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
#ifdef GPUGLM_FASTMATH
		output[i] = __logf(input[i]);
#else
		output[i] = log(input[i]);
#endif
	}
	return;
}
__global__ void cudaRecip(int n, num_t *input, num_t *output) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
#ifdef GPUGLM_FASTMATH
		output[i] = __frcp_rn(input[i]);
#else
		output[i] = 1.0 / input[i];
#endif
	}
	return;
}
__global__ void cudaSqRecip(int n, num_t *input, num_t *output) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
#ifdef GPUGLM_FASTMATH
		output[i] = __frcp_rn(__fmul_rn(input[i], input[i]));
#else
		output[i] = 1.0 / (input[i] * input[i]);
#endif
	}
	return;
}
__global__ void cudaNegBin(int n, num_t *input, num_t *output, num_t k) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
#ifdef GPUGLM_FASTMATH
		output[i] = __logf(__fdividef(input[i], __fadd_rn(input[i], k)));
#else
		output[i] = log(input[i] / (k + input[i]));
#endif
	}
	return;
}

// Inverse Link Functions /////////////////////////////////////////////////////

__global__ void cudaInvLogit(int n, num_t *input, num_t *output) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
#ifdef GPUGLM_FASTMATH
		output[i] = __fdividef(1.0, __fadd_rn(1.0, __expf(-input[i])));
#else
		output[i] = 1.0 / (1.0 + exp(-input[i]));
#endif
	}
	return;
}
__global__ void cudaExp(int n, num_t *input, num_t *output) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
#ifdef GPUGLM_FASTMATH
		output[i] = __expf(input[i]);
#else
		output[i] = exp(input[i]);
#endif
	}
	return;
}
__global__ void cudaSqrtRecip(int n, num_t *input, num_t *output) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
#ifdef GPUGLM_FASTMATH
		output[i] = __fdividef(1.0, __fsqrt_rn(input[i]));
#else
		output[i] = 1.0  / sqrt(input[i]);
#endif
	}
	return;
}
__global__ void cudaInvNegBin(int n, num_t *input, num_t *output, num_t k) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float p;
	if (i < n) {
#ifdef GPUGLM_FASTMATH
		p = __expf(input[i]);
		output[i] = __fdividef(__fmul_rn(p, k), __fsub_rn(1.0, p));
#else
		p = exp(input[i]);
		output[i] = (p * k) / (1 - p);
#endif
	}
	return;
}

///////////////////////////////////////////////////////////////////////////////
// Host-side Link Functions                                                  //
///////////////////////////////////////////////////////////////////////////////

void sapply(glmVector<num_t> *input, glmVector<num_t> *output,
		void (*cudaKernel)(int, num_t*, num_t*)) {
	int numBlocks = input->getNumBlocks();

	(*cudaKernel)<<<numBlocks, THREADS_PER_BLOCK>>>(input->getLength(),
			input->getDeviceData(), output->getDeviceData());

	return;
}

void sapply(glmVector<num_t> *input, glmVector<num_t> *output,
		void (*cudaKernel)(int, num_t*, num_t*, num_t), num_t k) {
	int numBlocks = input->getNumBlocks();

	(*cudaKernel)<<<numBlocks, THREADS_PER_BLOCK>>>(input->getLength(),
			input->getDeviceData(), output->getDeviceData(), k);

	return;
}

void linkLogit(glmVector<num_t> *input, glmVector<num_t> *output, num_t k) {
	sapply(input, output, cudaLogit);
	return;
}

void linkLog(glmVector<num_t> *input, glmVector<num_t> *output, num_t k) {
	sapply(input, output, cudaLog);
	return;
}

void linkRecip(glmVector<num_t> *input, glmVector<num_t> *output, num_t k) {
	sapply(input, output, cudaRecip);
	return;
}

void linkSqRecip(glmVector<num_t> *input, glmVector<num_t> *output, num_t k) {
	sapply(input, output, cudaSqRecip);
	return;
}

void linkNegBin(glmVector<num_t> *input, glmVector<num_t> *output, num_t k) {
	sapply(input, output, cudaNegBin, k);
	return;
}

void linkIdentity(glmVector<num_t> *input, glmVector<num_t> *output, num_t k) {
	// The identity link doesn't require any computation, but we include a stub
	// for the link/inverse link function so that the application code doesn't
	// need any additional logic to handle this special case.
	return;
}

// Inverse Link Functions /////////////////////////////////////////////////////

void linkInvLogit(glmVector<num_t> *input, glmVector<num_t> *output, num_t k) {
	sapply(input, output, cudaInvLogit);
	return;
}

void linkExp(glmVector<num_t> *input, glmVector<num_t> *output, num_t k) {
	sapply(input, output, cudaExp);
	return;
}
void linkSqrtRecip(glmVector<num_t> *input, glmVector<num_t> *output, num_t k) {
	sapply(input, output, cudaSqrtRecip);
	return;
}

void linkInvNegBin(glmVector<num_t> *input, glmVector<num_t> *output, num_t k) {
	sapply(input, output, cudaInvNegBin, k);
	return;
}

///////////////////////////////////////////////////////////////////////////////
// Link and Inverse Link Function Generators                                 //
///////////////////////////////////////////////////////////////////////////////

linkFunction getLinkFunction(std::string linkType) {
	linkFunction link = linkIdentity;

	if (linkType == "log") {
		link = linkLog;
	} else if (linkType == "logit") {
		link = linkLogit;
	} else if (linkType == "reciprocal") {
		link = linkRecip;
	} else if (linkType == "squared reciprocal") {
		link = linkSqRecip;
	} else if (linkType == "negative binomial") {
		link = linkNegBin;
	}

	return link;
}

linkFunction getInvLinkFunction(std::string linkType) {
	linkFunction link = linkIdentity;

	if (linkType == "log") {
		link = linkExp;
	} else if (linkType == "logit") {
		link = linkInvLogit;
	} else if (linkType == "reciprocal") {
		link = linkRecip;
	} else if (linkType == "squared reciprocal") {
		link = linkSqrtRecip;
	} else if (linkType == "negative binomial") {
		link = linkInvNegBin;
	}

	return link;
}
